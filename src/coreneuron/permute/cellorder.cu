#include "hip/hip_runtime.h"
/*
# =============================================================================
# Copyright (c) 2016 - 2021 Blue Brain Project/EPFL
#
# See top-level LICENSE file for details.
# =============================================================================
*/

#include "coreneuron/utils/utils_cuda.h"
#include "coreneuron/permute/cellorder.hpp"
#include "coreneuron/network/tnode.hpp"
#include "coreneuron/sim/multicore.hpp"

namespace coreneuron {

__device__ void triang_interleaved2_device(NrnThread* nt,
                                           int icore,
                                           int ncycle,
                                           int* stride,
                                           int lastnode) {
    int icycle = ncycle - 1;
    int istride = stride[icycle];
    int i = lastnode - istride + icore;

    int ip;
    double p;
    while (icycle >= 0) {
        // most efficient if istride equal warpsize, else branch divergence!
        if (icore < istride) {
            ip = nt->_v_parent_index[i];
            p = nt->_actual_a[i] / nt->_actual_d[i];
            atomicAdd(&nt->_actual_d[ip], -p * nt->_actual_b[i]);
            atomicAdd(&nt->_actual_rhs[ip], -p * nt->_actual_rhs[i]);
        }
        --icycle;
        istride = stride[icycle];
        i -= istride;
    }
}

__device__ void bksub_interleaved2_device(NrnThread* nt,
                                          int root,
                                          int lastroot,
                                          int icore,
                                          int ncycle,
                                          int* stride,
                                          int firstnode) {
    for (int i = root; i < lastroot; i += warpsize) {
        nt->_actual_rhs[i] /= nt->_actual_d[i];  // the root
    }

    int i = firstnode + icore;

    int ip;
    for (int icycle = 0; icycle < ncycle; ++icycle) {
        int istride = stride[icycle];
        if (icore < istride) {
            ip = nt->_v_parent_index[i];
            nt->_actual_rhs[i] -= nt->_actual_b[i] * nt->_actual_rhs[ip];
            nt->_actual_rhs[i] /= nt->_actual_d[i];
        }
        i += istride;
    }
}

__global__ void solve_interleaved2_kernel(NrnThread* nt, InterleaveInfo* ii, int ncore) {
    int icore = blockDim.x * blockIdx.x + threadIdx.x;

    int* ncycles = ii->cellsize;         // nwarp of these
    int* stridedispl = ii->stridedispl;  // nwarp+1 of these
    int* strides = ii->stride;           // sum ncycles of these (bad since ncompart/warpsize)
    int* rootbegin = ii->firstnode;      // nwarp+1 of these
    int* nodebegin = ii->lastnode;       // nwarp+1 of these

    while (icore < ncore) {
        int iwarp = icore / warpsize;     // figure out the >> value
        int ic = icore & (warpsize - 1);  // figure out the & mask
        int ncycle = ncycles[iwarp];
        int* stride = strides + stridedispl[iwarp];
        int root = rootbegin[iwarp];
        int lastroot = rootbegin[iwarp + 1];
        int firstnode = nodebegin[iwarp];
        int lastnode = nodebegin[iwarp + 1];

        triang_interleaved2_device(nt, ic, ncycle, stride, lastnode);
        bksub_interleaved2_device(nt, root + ic, lastroot, ic, ncycle, stride, firstnode);

        icore += blockDim.x * gridDim.x;
    }
}

void solve_interleaved2_launcher(NrnThread* nt, InterleaveInfo* info, int ncore, void* stream) {
    auto hip_stream = static_cast<hipStream_t>(stream);

    /// the selection of these parameters has been done after running the channel-benchmark for
    /// typical production runs, i.e. 1 MPI task with 1440 cells & 6 MPI tasks with 8800 cells.
    /// In the OpenACC/OpenMP implementations threadsPerBlock is set to 32. From profiling the
    /// channel-benchmark circuits mentioned above we figured out that the best performance was
    /// achieved with this configuration
    int threadsPerBlock = warpsize;
    /// Max number of blocksPerGrid for NVIDIA GPUs is 65535, so we need to make sure that the
    /// blocksPerGrid we launch the CUDA kernel with doesn't exceed this number
    const auto maxBlocksPerGrid = 65535;
    int provisionalBlocksPerGrid = (ncore + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid = provisionalBlocksPerGrid <= maxBlocksPerGrid ? provisionalBlocksPerGrid
                                                                     : maxBlocksPerGrid;

    solve_interleaved2_kernel<<<blocksPerGrid, threadsPerBlock, 0, hip_stream>>>(nt, info, ncore);

    hipStreamSynchronize(hip_stream);

    CHECKLAST("solve_interleaved2_launcher");
}

}  // namespace coreneuron
